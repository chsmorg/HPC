#include "hip/hip_runtime.h"
using namespace std;
#include <stdio.h>
#include <iostream>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "helper_image.h"
#include <thrust/sort.h>
#define N 10

void medianFilterCPU(float* image, int width, int height, int windowSize, int x, int y, float* output) {

    // Skip the pixel if it is outside the image bounds
    if (x >= width || y >= height) return;

    // Create a list of the neighboring pixel values
     const int size = windowSize * windowSize;
    float* values = new float[size];
        
    int index = 0;
    for (int i = -windowSize/2; i <= windowSize/2; i++) {
        for (int j = -windowSize/2; j <= windowSize/2; j++) {
                // Compute the coordinates of the neighboring pixel
                int xn = x + i;
                int yn = y + j;
                // Skip the pixel if it is outside the image bounds
                if (xn < 0 || xn >= width || yn < 0 || yn >= height) continue;
                // Add the value of the neighboring pixel to the list
                values[index] = image[(yn * width + xn)];
                index++;
        }
    }

    // Sort the list of neighboring pixel values
    
    for (int j = 0; j < size; j++) {
        for (int k = 0; k < size - j - 1; k++) {
            if (values[k] > values[k + 1]) {
                float temp = values[k];
                values[k] = values[k + 1];
                values[k + 1] = temp;
            }
        }
    }
    output[(y * width + x)] = values[index/2];
    delete values;
}


__global__ void medianFilter(float* image, int width, int height, int windowSize, float* output) {
    // Compute the coordinates of the current pixel
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    //output[(y * width + x)] = y;

    // Skip the pixel if it is outside the image bounds
    if (x >= width || y >= height) return;

    // Create a list of the neighboring pixel values
     const int size = windowSize * windowSize;
    // float values[size];
    float* values = new float[size];
        
    int index = 0;
    for (int i = -windowSize/2; i <= windowSize/2; i++) {
        for (int j = -windowSize/2; j <= windowSize/2; j++) {
                // Compute the coordinates of the neighboring pixel
                int xn = x + i;
                int yn = y + j;
                // Skip the pixel if it is outside the image bounds
                if (xn < 0 || xn >= width || yn < 0 || yn >= height) continue;
                // Add the value of the neighboring pixel to the list
                values[index] = image[(yn * width + xn)];
                index++;
        }
    }
    
    //Sort the list of neighboring pixel values
    for (int j = 0; j < index; j++) {
        for (int k = 0; k < index - j - 1; k++) {
            if (values[k] > values[k + 1]) {
                float temp = values[k];
                values[k] = values[k + 1];
                values[k + 1] = temp;
            }
        }
    }
    
    output[(y * width + x)] = values[index/2];
    delete values;
}

float * gpuFilter(int filter, char* infile,char* outfile){
    printf("Start Gpu\n");
    float *imageData = NULL;
    unsigned int width, height;
    sdkLoadPGM(infile, &imageData, &width, &height);
    
    //Allocate memory for the input and output image on the device
    float* d_image;
    float* d_imageOut;
    hipMalloc((void**)&d_image, width * height * sizeof(float));
    hipMalloc((void**)&d_imageOut, width * height * sizeof(float));
    
    cudaThreadSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
    
        // Copy the input image from the host to the device
    hipMemcpy(d_image, imageData, width * height  * sizeof(float), hipMemcpyHostToDevice);
    
    //dim3 threads(16, 16);
    //dim3 blocks(512/threads.x, 512/threads.y);
    
    dim3 threads32(32,32);
    dim3 blocks32(32, 32);
    
    auto start = chrono::high_resolution_clock::now();
    medianFilter<<<blocks32, threads32>>>(d_image, width, height, filter, d_imageOut);
    //medianFilter<<<blocks, threads>>>(d_image, width, height, filter, d_imageOut);
    
    hipDeviceSynchronize();
    
        auto stop = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start);

        std::cout << "Time taken to apply median filter on GPU: "
                  << duration.count() << " milliseconds" << std::endl;

    
    
    
    
    
    
    
    hipMemcpy(imageData, d_imageOut, width * height * sizeof(float), hipMemcpyDeviceToHost);
    sdkSavePGM(outfile, imageData, width, height);
    hipFree(d_image);
    return imageData;
    
}

float* cpuFilter(int filter, char* infile, char* outfile){
    printf("Start Cpu\n");
    float *imageData = NULL;
    unsigned int width, height;
    sdkLoadPGM(infile, &imageData, &width, &height);
    float *outData = new float[height*width];
    auto start = chrono::high_resolution_clock::now();
    for(int i = 0; i < width; i++){
        for(int j = 0; j<height; j++){
            medianFilterCPU(imageData,width,height,filter,i,j,outData);
        }
    }
    auto stop = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start);

        std::cout << "Time taken to apply median filter on CPU: "
                  << duration.count() << " milliseconds" << std::endl;
    sdkSavePGM(outfile, outData, width, height);
    return outData;
}


float cpc(float*cpu, float*gpu){
    float total_pixels = 512*512;
    // cout<<"total_pixels: "<<total_pixels<<endl;
    float not_matching_pixels = 0.0;
    for (int i = 0; i < 512; i++) {
        //cout<<"Row: "<<i<<endl;
        for (int j = 0; j < 512; j++) {
            if(gpu[i * 512 + j] != cpu[i * 512 + j]){
                not_matching_pixels += 1.0;
            }
        }
    }
    float percent_correctness = (total_pixels - not_matching_pixels) / (total_pixels) * 100;
    return percent_correctness;
}


int main(int argc, char* argv[]){
    
    int filter = atoi(argv[1]);
    float* gpu = gpuFilter(filter, argv[2],argv[3]);
    float* cpu = cpuFilter(filter,argv[2],argv[3]);
    std::cout<<cpc(cpu,gpu)<<std::endl;

}