#include "hip/hip_runtime.h"
using namespace std;
#include "hw6.h"

void print_arrayAt(Displacement* arr, int col, int gens){
    
   int j = col*gens;
    
    for(int i = 0; i <gens; i++){
        int k = j+i;
        std::cout<<"gen: "<<arr[k].gen<<"  col: "<<arr[k].col<<"  theta: "<<arr[k].theta<<"  x: "<<arr[k].x<<"  y: "<<arr[k].y<<std::endl;
    }
    return;
    
}

Displacement* serialCPU(int N, int gens, FILE *fp){
        std::vector<Displacement*> vec1 = readDisplacementData(fp,N);
        std::vector<std::pair<int,int>> blocked = createBlocksList(vec1,N);
        Displacement* m = createArray(vec1,N,gens);
        Block *b = createBlocks(blocked);
        int block_size = blocked.size();
    
        auto start = chrono::high_resolution_clock::now();
        std::cout<<"starting serial CPU Growth..."<<std::endl;
    
        growCpu(m,N,gens,b,block_size);
    
        auto stop = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start);
    
        std::cout << "Time taken to Grow Serial on CPU: "
                  << duration.count() << " Milliseconds" << std::endl;
        
        free(b);
        fclose(fp);
        writeTreeToCsvList(m,N,gens);
    
       // print_arrayAt(m,50,gens);
        
        return m;
    
    
}


Displacement* serialGPU(int N, int gens,FILE *fp){
    
    
        std::vector<Displacement*> vec1 = readDisplacementData(fp,N);
        std::vector<std::pair<int,int>> blocked = createBlocksList(vec1,N);
        Block *b = createBlocks(blocked);
        int block_size = blocked.size();
    
        Displacement* array;
        Block* block_array;
    
    
        Displacement* m = createArray(vec1,N,gens);
    
        hipMalloc((void**)&array, sizeof(Displacement)*gens*N);
        hipMalloc((void**)&block_array, sizeof(Block)*block_size);
    
    
        hipMemcpy(array, m, sizeof(Displacement)*gens*N, hipMemcpyHostToDevice);
        hipMemcpy(block_array, b, sizeof(Block)*block_size, hipMemcpyHostToDevice);
    
        dim3 threads(1, 1);
        dim3 blocks(block_size, 1);
    
    
        auto start = chrono::high_resolution_clock::now();
        std::cout<<"starting serial GPU Growth..."<<std::endl;
        grow<<<blocks, threads>>>(array, N, gens, block_array, block_size);
    
        hipDeviceSynchronize();
    
        auto stop = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start);

        std::cout << "Time taken to Grow Serial on GPU: "
                  << duration.count() << " Milliseconds" << std::endl;
        hipMemcpy(m, array, sizeof(Displacement)*gens*N, hipMemcpyDeviceToHost);
    
        //write serial to output file:
    
        //writeTreeToCsvList(m,N,gens);
    
        
    
    free(b);
    hipFree(array);
    hipFree(block_array);
    fclose(fp);
    return m;
    
    
}

int* locateIndexes(std::vector<Displacement*> vec1, int N, int gens, Displacement* m){
    
        std::vector<std::pair<int,int>> blocked = createBlocksList(vec1,N);
        Block *b = createBlocks(blocked);
        Displacement* array;
        Block* block_array;
        int* indexes;
    
        int temp_gens = 10;
        int block_size = blocked.size();
        int* col_indexes = (int*)malloc(sizeof(int)*N);
        hipMalloc((void**)&array, sizeof(Displacement)*gens*N);
        hipMalloc((void**)&block_array, sizeof(Block)*block_size);
        hipMalloc((void**)&indexes, sizeof(int)*N);
    
        hipMemcpy(array, m, sizeof(Displacement)*gens*N, hipMemcpyHostToDevice);
        hipMemcpy(block_array, b, sizeof(Block)*block_size, hipMemcpyHostToDevice);
    
        dim3 blocks(1, 1);
        dim3 grid(block_size, 1);
        auto start = chrono::high_resolution_clock::now();
    
        findIndexes<<<grid, blocks>>>(array, N, temp_gens, block_array, block_size,indexes);
    
        hipDeviceSynchronize();
        auto stop = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start);

        std::cout << "Time taken to Find Indexes on GPU: "
                  << duration.count() << " Milliseconds" << std::endl;
        hipMemcpy(col_indexes, indexes, sizeof(int)*N, hipMemcpyDeviceToHost);
    
        hipFree(array);
        hipFree(block_array);
        hipFree(indexes);
        free(b);

        return col_indexes;
    
        
    
}

Displacement* parallelGPU(int N, int gens, int P, FILE *fp){
    
        std::vector<Displacement*> vec1 = readDisplacementData(fp,N);
        
    
        Displacement* array;
        int* indexes;
    
         Displacement* m = createArray(vec1,N,gens);
         int* col_indexes = locateIndexes(vec1,N,gens,m);
        hipMalloc((void**)&array, sizeof(Displacement)*gens*N);
      
        hipMalloc((void**)&indexes, sizeof(int)*N);
        
    
    
        hipMemcpy(array, m, sizeof(Displacement)*gens*N, hipMemcpyHostToDevice);
  
        hipMemcpy(indexes, col_indexes, sizeof(int)*N, hipMemcpyHostToDevice);
 
        
    
        
                            
 
    

    
    
        std::cout<<"starting Parallel GPU Growth..."<<std::endl;
        auto start = chrono::high_resolution_clock::now();

    
    if(P == 1){
                dim3 gridDim(N,gens/N);
                dim3 blockDim(1,N);
                growFullTree<<<gridDim,blockDim>>>(array,N,gens,indexes); 
        }
    else if(P == 2){
                dim3 gridDim(N,N);
                dim3 blockDim(1,gens/N);
                growFullTree<<<gridDim,blockDim>>>(array,N,gens,indexes); 
        }
    else if(P==3){
                dim3 gridDim(N,N/2);
                dim3 blockDim(1,round(gens/(N/2)));
                growFullTree<<<gridDim,blockDim>>>(array,N,gens,indexes); 
    }
    else if(P==4){
                dim3 gridDim(N,N/4);
                dim3 blockDim(1,round(gens/(N/4)));
                growFullTree<<<gridDim,blockDim>>>(array,N,gens,indexes); 
    }
    else{
                std::cout<<"invalid Dim size, range 1-4. look at readMe for more info. Dim 1 being used"<<std::endl;
                dim3 gridDim(N,gens/N);
                dim3 blockDim(1,N);
                growFullTree<<<gridDim,blockDim>>>(array,N,gens,indexes); 
    }
         
        hipDeviceSynchronize();
    
        auto stop = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::nanoseconds>(stop - start);

        std::cout << "Time taken to Grow Parallel on GPU: "
                  << duration.count() << " Nanoseconds" << std::endl;
    
        hipMemcpy(m, array, sizeof(Displacement)*gens*N, hipMemcpyDeviceToHost);
    
        writeTreeToCsvList(m,N,gens);
    
        //print_arrayAt(m,50,gens);
    
   
    
        
        free(col_indexes);
        hipFree(array);
        hipFree(indexes);
        fclose(fp);
    return m;
}


void compare(Displacement* arr1, Displacement* arr2, int N, int gens){
    float not_matching = 0;
    
    for(int i = 0; i<N; i++){
        for(int j = 0; j<gens; j++){
            if(arr1[i*gens +j].x != arr2[i*gens+j].x){
                not_matching += 1;
            }
        }
    }
    float p = ((gens*N)-not_matching) / (gens*N) * 100;
    std::cout<<"total matching: "<<p<<"%"<<std::endl;
    
}



int main(int argc, char* argv[]){
    int N = atoi(argv[2]);
    int gens = atoi(argv[3]);
    int P = atoi(argv[4]);
    
    FILE *fp;
    fp = fopen(argv[1],"r");
    if(!fp){
        printf("File not found\n");
        exit(0);
    }
    //this the fast one
    Displacement* gpu_fast = parallelGPU(N,gens,P,fp);
    fp = fopen(argv[1],"r");
    //this the slow one
    Displacement* gpu = serialGPU(N,gens,fp);
    fp = fopen(argv[1],"r");
    
    //this the normal one
     Displacement* cpu = serialCPU(N,gens,fp);
    std::cout<<"compare for faster gpu growth: "<<std::endl;
    compare(cpu,gpu_fast,N,gens);
    std::cout<<"compare for slower gpu growth: "<<std::endl;
    compare(cpu,gpu,N,gens);
    
    free(gpu);
    free(cpu);
    free(gpu_fast);

}
