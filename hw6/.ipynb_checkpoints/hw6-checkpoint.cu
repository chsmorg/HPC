#include "hip/hip_runtime.h"
#include "hw6.h"
using namespace std::chrono;





__device__ bool checkRight(Displacement * nodes, int gens, int col, int index){

    double x1 = nodes[gens*col].x;
    // Displacement *node = (nodes+gens*col);
    for(int i = 0; i<=index; i++){
        if(abs((x1) - (nodes[gens*(col+1)+i].x)) < 5e-08){
            nodes[gens*col+1].theta = 90;
            nodes[gens*col].x = nodes[gens*col+1].x;
            nodes[gens*(col+1)+1].theta = 90;
            nodes[gens*(col+1)].x = nodes[gens*(col+1)+1].x;
            return true;
        }
    }
    return false;


}


__device__ bool checkLeft(Displacement * nodes, int gens, int col, int index){

    double x1 = nodes[gens*col].x;

    for(int i = 0; i<=index; i++){
        if(abs((x1) - (nodes[gens*(col-1)+i].x)) < 5e-08){
            nodes[gens*col+1].theta = 90;
            nodes[gens*col].x = nodes[gens*col+1].x;
            nodes[gens*(col-1)+1].theta = 90;
            nodes[gens*(col-1)].x = nodes[gens*(col-1)+1].x;
            return true;
        }
    }
    return false;


}


__device__ void propagate(Displacement* nodes, double x, double y, double theta, int index,int col){

    for(int i = index-1; i > -1; i--){
        if(theta != 90){
            nodes[col+i].x += x;
        }
        nodes[col+i].y += y;

    }
}

__device__ void  growCol(Displacement * nodes, int col, int index){
    //create new node and assign values then add to front of col

    double y = nodes[col+index-2].y;
    double x = nodes[col+index-2].x-nodes[col+index-1].x;
    double theta = nodes[col+1].theta;

    nodes[col+index].col = nodes[col+index-1].col;

    nodes[col+index].gen = nodes[col+index-1].gen+1;
    nodes[col+index].y = 0;
    nodes[col+index].theta = theta;
    nodes[col+index].x = nodes[col+index-1].x;

    propagate(nodes,x,y,theta, index,col);

    return;
}

__global__ void grow(Displacement* tree, int N, int generations, Block* blocks, int block_size){
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(x < block_size){
        int start = blocks[x].start;
        int end = blocks[x].end;
        
        
        for(unsigned int i = 7; i < generations; i++){
        for(unsigned int j = start; j < end ; j++) growCol(tree, generations*j,i);
        
        for(unsigned int j = start; j < end ; j++){
            if(tree[generations*j+i-1].theta < 90 && j < end-1){
                checkRight(tree, generations,j,i);
            }
            else if( j > start && tree[generations*j+i-1].theta > 90) {
                checkLeft(tree, generations,j,i);
            }
            
        }
    }
    }
    
}

__global__ void findIndexes(Displacement* tree, int N, int generations, Block* blocks, int block_size, int* indexes){
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(x < block_size){
        int start = blocks[x].start;
        int end = blocks[x].end;
        int range = end-start;
        int index = 0;
        for(unsigned int i = 7; i < generations; i++){
        for(unsigned int j = start; j < end ; j++) growCol(tree, generations*j,i);

        for(unsigned int j = start; j < end ; j++){
            
            if(indexes[j] != 0) continue;
            bool col = false;
            if(tree[generations*j+i-1].theta < 90 && j < end-1){
                col = checkRight(tree, generations,j,i);
            }
            else if( j > start && tree[generations*j+i-1].theta > 90) {
                col =checkLeft(tree, generations,j,i);
            }
            if(j ==0 || j == N-1){
                indexes[j] = -1;
                index++;
            }
            
            if(col){
                indexes[j] = i;
                index++;
            }
                
        }
          if(index == range) break;
    }
    }
    
}

__global__ void growFullTree(Displacement* tree, int N, int generations, int* indexes){
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(x < N && y < generations && y > 6){
        int in = generations*x+y;
        double theta;
         double dx = tree[generations*x].x - tree[generations*x+1].x;
        if(x>indexes[x]){
            theta = 90;
            tree[in].x = tree[generations*x].x + indexes[x]*dx;
        }
        else if(x<indexes[x] || indexes[x]<0){
           tree[in].x = tree[generations*x].x + (y*dx);
           theta = tree[generations*x+5].theta;
           
       }
        double dy = tree[generations*x+5].y;
        tree[in].theta = theta;
       
       tree[in].y = dy*(y);
       tree[in].col = x;
       tree[in].gen = generations-y+6;
    }
    //tree[generations*x+y].col = x;
    //tree[generations*x+y].gen = y;
   }
  


